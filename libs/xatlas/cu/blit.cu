#include <libgpu/cuda/cu/opencl_translator.cu>

#include "../cl/blit.cl"
#include "libgpu/work_size.h"

void cuda_blit(const gpu::WorkSize &workSize,
			   const unsigned long *atlases, const unsigned long *charts,
			   const int limit_w,
			   const int limit_h,
			   const int levels,
			   const int rate,
			   const unsigned int *atlasSizes,
			   const unsigned int *chartSizes,
			   unsigned int *best,
			   unsigned char *results,
			   hipStream_t stream)
{
	blit<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(atlases, charts,
																	   limit_w, limit_h,
																	   levels,
																	   rate,
																	   atlasSizes,
																	   chartSizes,
																	   best,
																	   results);
	CUDA_CHECK_KERNEL(stream);
}

void cuda_blitLevel(const gpu::WorkSize &workSize,
					const unsigned long *atlases,
					const unsigned long *charts,
					const int w, const int h,
					const int limit_w,
					const int limit_h,
					const int level,
					const int rate,
					const unsigned int *atlasSizes,
					const unsigned int *chartSizes,
					unsigned int *best_metric,
					unsigned int *results,
					hipStream_t stream)
{
	blitLevel<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(atlases, charts,
																			w, h,
																			limit_w, limit_h,
																			level,
																			rate,
																			atlasSizes,
																			chartSizes,
																			false,
																			0,
																			nullptr,
																			nullptr,
																			best_metric,
																			results);
}


void cuda_blitFiltered(const gpu::WorkSize &workSize,
					   const unsigned long *atlases,
					   const unsigned long *charts,
					   const int w, const int h,
					   const int limit_w,
					   const int limit_h,
					   const int level,
					   const int rate,
					   const unsigned int *atlasSizes,
					   const unsigned int *chartSizes,
					   const bool filtered,
					   const unsigned int candidates_num,
					   const unsigned short *filter_x,
					   const unsigned short *filter_y,
					   unsigned int *best_metric,
					   unsigned int *results,
					   hipStream_t stream)
{
	blitLevel<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(atlases, charts,
																			w, h,
																			limit_w, limit_h,
																			level,
																			rate,
																			atlasSizes,
																			chartSizes,
																			filtered,
																			candidates_num,
																			filter_x,
																			filter_y,
																			best_metric,
																			results);
}

void cuda_bufferCleanup(const gpu::WorkSize &workSize,
					 unsigned int *array,
					 const uint64_t n,
					 hipStream_t stream)
{
	bufferCleanup<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(array, n);
}

void cuda_scanReduce(const gpu::WorkSize &workSize,
					 unsigned int *array,
					 const uint64_t limit,
					 const uint64_t offset,
					 hipStream_t stream)
{
	scanReduce<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(array, limit, offset);
}

void cuda_scanDownSweep(const gpu::WorkSize &workSize,
						unsigned int *array,
						const uint64_t limit,
						const uint64_t offset,
						hipStream_t stream)
{
	scanDownSweep<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(array, limit, offset);
}


void cuda_aggregateResults(const gpu::WorkSize &workSize,
						   const unsigned int *array,
						   const int limit_w, const int limit_h,
						   unsigned short *results_x,
						   unsigned short *results_y,
						   hipStream_t stream)
{
	aggregateResults<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(array, limit_w, limit_h, results_x, results_y);
}